#include "hip/hip_runtime.h"
#include "data.cuh"
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(err)); \
        exit(1); \
    } \
}

// Hyperparameters
#define LEARNING_RATE 1e-3f
#define BATCH_SIZE 32
#define N_CONV_LAYERS 3
#define N_FILTERS 64
#define KERNEL_SIZE 3
#define EPSILON 1e-5f
#define MAX_GRAD_NORM 1.0f

typedef struct {
    float *weights;
    float *bias;
    
    // Temporary storage for backprop
    float *d_weights;
    float *d_bias;
} ConvLayer;

typedef struct {
    float *weights;
    float *bias;
    
    // Temporary storage for backprop
    float *d_weights;
    float *d_bias;
} DenseLayer;

typedef struct {
    ConvLayer *conv_layers;
    DenseLayer dense_layer;
    int n_conv_layers;
    int kernel_size;
    int n_filters;
    int sequence_length;
    int n_inputs;
    int n_outputs;
} Model;

__global__ void init_weights_kernel(float *weights, int size, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(clock64(), idx, 0, &state);
        weights[idx] = (hiprand_uniform(&state) - 0.5f) * scale;
    }
}

static Model* create_model(int sequence_length, int n_inputs, int n_outputs) {
    Model *model = (Model*)malloc(sizeof(Model));
    model->n_conv_layers = N_CONV_LAYERS;
    model->kernel_size = KERNEL_SIZE;
    model->n_filters = N_FILTERS;
    model->sequence_length = sequence_length;
    model->n_inputs = n_inputs;
    model->n_outputs = n_outputs;
    
    // Allocate conv layers
    model->conv_layers = (ConvLayer*)malloc(N_CONV_LAYERS * sizeof(ConvLayer));
    
    for (int i = 0; i < N_CONV_LAYERS; i++) {
        int in_channels = (i == 0) ? n_inputs : N_FILTERS;
        int weights_size = N_FILTERS * in_channels * KERNEL_SIZE;
        
        // Allocate layer parameters
        CHECK_CUDA(hipMalloc(&model->conv_layers[i].weights, weights_size * sizeof(float)));
        CHECK_CUDA(hipMalloc(&model->conv_layers[i].bias, N_FILTERS * sizeof(float)));
        
        // Allocate gradients
        CHECK_CUDA(hipMalloc(&model->conv_layers[i].d_weights, weights_size * sizeof(float)));
        CHECK_CUDA(hipMalloc(&model->conv_layers[i].d_bias, N_FILTERS * sizeof(float)));
        
        // Initialize weights
        float scale = sqrtf(2.0f / (in_channels * KERNEL_SIZE)); // He initialization
        init_weights_kernel<<<(weights_size + 255) / 256, 256>>>(
            model->conv_layers[i].weights, weights_size, scale);
        
        // Initialize bias
        CHECK_CUDA(hipMemset(model->conv_layers[i].bias, 0, N_FILTERS * sizeof(float)));
    }
    
    // Initialize dense layer
    int dense_weights_size = N_FILTERS * n_outputs;
    CHECK_CUDA(hipMalloc(&model->dense_layer.weights, dense_weights_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&model->dense_layer.bias, n_outputs * sizeof(float)));
    CHECK_CUDA(hipMalloc(&model->dense_layer.d_weights, dense_weights_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&model->dense_layer.d_bias, n_outputs * sizeof(float)));
    
    float dense_scale = sqrtf(2.0f / N_FILTERS);
    init_weights_kernel<<<(dense_weights_size + 255) / 256, 256>>>(
        model->dense_layer.weights, dense_weights_size, dense_scale);
    CHECK_CUDA(hipMemset(model->dense_layer.bias, 0, n_outputs * sizeof(float)));
    
    return model;
}

// Structure to hold intermediate activations
typedef struct {
    float *conv_inputs;     // Input to each conv layer
    float *conv_outputs;    // Output after conv
    float *norm_outputs;    // Output after RMSNorm
    float *relu_outputs;    // Output after ReLU
    float *residual_outputs; // Residual connections
    float *pooled_output;   // After global average pooling
    float *final_output;    // Network output
} Activations;

static Activations* create_activations(Model *model, int batch_size) {
    Activations *acts = (Activations*)malloc(sizeof(Activations));
    int seq_features = batch_size * model->sequence_length * model->n_filters;
    int input_features = batch_size * model->sequence_length * model->n_inputs;
    
    CHECK_CUDA(hipMalloc(&acts->conv_inputs, input_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&acts->conv_outputs, seq_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&acts->norm_outputs, seq_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&acts->relu_outputs, seq_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&acts->residual_outputs, seq_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&acts->pooled_output, 
        batch_size * model->n_filters * sizeof(float)));
    CHECK_CUDA(hipMalloc(&acts->final_output, 
        batch_size * model->n_outputs * sizeof(float)));
    
    return acts;
}

static void free_activations(Activations *acts) {
    CHECK_CUDA(hipFree(acts->conv_inputs));
    CHECK_CUDA(hipFree(acts->conv_outputs));
    CHECK_CUDA(hipFree(acts->norm_outputs));
    CHECK_CUDA(hipFree(acts->relu_outputs));
    CHECK_CUDA(hipFree(acts->residual_outputs));
    CHECK_CUDA(hipFree(acts->pooled_output));
    CHECK_CUDA(hipFree(acts->final_output));
    free(acts);
}

__global__ void conv1d_forward_kernel(
    const float *input, float *output,
    const float *weights, const float *bias,
    int batch_size, int sequence_length, int n_inputs, 
    int n_filters, int kernel_size
) {
    int batch_idx = blockIdx.x;
    int filter_idx = blockIdx.y;
    int seq_idx = threadIdx.x;
    
    if (seq_idx < sequence_length) {
        float sum = bias[filter_idx];
        
        for (int k = 0; k < kernel_size; k++) {
            int seq_pos = seq_idx - kernel_size/2 + k;
            if (seq_pos >= 0 && seq_pos < sequence_length) {
                for (int c = 0; c < n_inputs; c++) {
                    float input_val = input[
                        batch_idx * sequence_length * n_inputs + 
                        seq_pos * n_inputs + c
                    ];
                    float weight = weights[
                        filter_idx * n_inputs * kernel_size + 
                        c * kernel_size + k
                    ];
                    sum += input_val * weight;
                }
            }
        }
        
        output[
            batch_idx * sequence_length * n_filters + 
            seq_idx * n_filters + filter_idx
        ] = sum;
    }
}

__global__ void rms_norm_forward_kernel(
    float *input, float *output,
    int batch_size, int sequence_length, int n_filters
) {
    int batch_idx = blockIdx.x;
    int seq_idx = threadIdx.x;
    
    if (seq_idx < sequence_length) {
        // Compute RMS for this position
        float sum_squared = 0.0f;
        for (int f = 0; f < n_filters; f++) {
            int idx = batch_idx * sequence_length * n_filters + 
                     seq_idx * n_filters + f;
            float val = input[idx];
            sum_squared += val * val;
        }
        
        float rms = sqrtf(sum_squared / n_filters + EPSILON);
        
        // Normalize using RMS
        for (int f = 0; f < n_filters; f++) {
            int idx = batch_idx * sequence_length * n_filters + 
                     seq_idx * n_filters + f;
            output[idx] = input[idx] / rms;
        }
    }
}

__global__ void relu_forward_kernel(
    float *input, float *output,
    int batch_size, int sequence_length, int n_filters
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * sequence_length * n_filters;
    
    if (idx < total_elements) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

__global__ void residual_add_kernel(
    float *input, float *residual,
    int batch_size, int sequence_length, int n_filters
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * sequence_length * n_filters;
    
    if (idx < total_elements) {
        input[idx] += residual[idx];
    }
}

__global__ void global_avg_pool_kernel(
    const float *input, float *output,
    int batch_size, int sequence_length, int n_filters
) {
    int batch_idx = blockIdx.x;
    int filter_idx = threadIdx.x;
    
    if (filter_idx < n_filters) {
        float sum = 0.0f;
        for (int s = 0; s < sequence_length; s++) {
            sum += input[
                batch_idx * sequence_length * n_filters + 
                s * n_filters + filter_idx
            ];
        }
        output[batch_idx * n_filters + filter_idx] = sum / sequence_length;
    }
}

__global__ void dense_forward_kernel(
    const float *input, float *output,
    const float *weights, const float *bias,
    int batch_size, int n_filters, int n_outputs
) {
    int batch_idx = blockIdx.x;
    int output_idx = threadIdx.x;
    
    if (output_idx < n_outputs) {
        float sum = bias[output_idx];
        for (int f = 0; f < n_filters; f++) {
            sum += input[batch_idx * n_filters + f] * 
                   weights[f * n_outputs + output_idx];
        }
        output[batch_idx * n_outputs + output_idx] = sum;
    }
}

typedef struct {
    float *d_conv_inputs;
    float *d_conv_outputs;
    float *d_norm_outputs;
    float *d_relu_outputs;
    float *d_pooled_output;
    float *d_final_output;
} Gradients;

static Gradients* create_gradients(Model *model, int batch_size) {
    Gradients *grads = (Gradients*)malloc(sizeof(Gradients));
    int seq_features = batch_size * model->sequence_length * model->n_filters;
    int input_features = batch_size * model->sequence_length * model->n_inputs;
    
    CHECK_CUDA(hipMalloc(&grads->d_conv_inputs, input_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&grads->d_conv_outputs, seq_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&grads->d_norm_outputs, seq_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&grads->d_relu_outputs, seq_features * sizeof(float)));
    CHECK_CUDA(hipMalloc(&grads->d_pooled_output, 
        batch_size * model->n_filters * sizeof(float)));
    CHECK_CUDA(hipMalloc(&grads->d_final_output, 
        batch_size * model->n_outputs * sizeof(float)));
    
    return grads;
}

__global__ void rms_norm_backward_kernel(
    const float *d_output, const float *input,
    float *d_input,
    int batch_size, int sequence_length, int n_filters
) {
    int batch_idx = blockIdx.x;
    int seq_idx = threadIdx.x;
    
    if (seq_idx < sequence_length) {
        // Compute RMS for this position
        float sum_squared = 0.0f;
        for (int f = 0; f < n_filters; f++) {
            int idx = batch_idx * sequence_length * n_filters + 
                     seq_idx * n_filters + f;
            float val = input[idx];
            sum_squared += val * val;
        }
        float rms = sqrtf(sum_squared / n_filters + EPSILON);
        float inv_rms = 1.0f / rms;
        
        // Compute gradients
        for (int f = 0; f < n_filters; f++) {
            int idx = batch_idx * sequence_length * n_filters + 
                     seq_idx * n_filters + f;
            float x_i = input[idx];
            float dy_i = d_output[idx];
            
            // Gradient for input considering the RMS normalization
            float dx = inv_rms * (dy_i - 
                (x_i * inv_rms / n_filters) * 
                (x_i * dy_i * inv_rms));
            
            d_input[idx] = dx;
        }
    }
}

__global__ void dense_backward_kernel(
    const float *d_output,
    const float *input,
    const float *weights,
    float *d_input,
    float *d_weights,
    float *d_bias,
    int batch_size,
    int n_filters,
    int n_outputs
) {
    int batch_idx = blockIdx.x;
    int filter_idx = threadIdx.x;
    
    if (filter_idx < n_filters) {
        float d_input_val = 0.0f;
        for (int o = 0; o < n_outputs; o++) {
            float d_output_val = d_output[batch_idx * n_outputs + o];
            d_input_val += d_output_val * weights[filter_idx * n_outputs + o];
            
            atomicAdd(&d_weights[filter_idx * n_outputs + o],
                     d_output_val * input[batch_idx * n_filters + filter_idx]);
        }
        d_input[batch_idx * n_filters + filter_idx] = d_input_val;
    }
    
    if (batch_idx == 0 && filter_idx < n_outputs) {
        float d_bias_val = 0.0f;
        for (int b = 0; b < batch_size; b++) {
            d_bias_val += d_output[b * n_outputs + filter_idx];
        }
        d_bias[filter_idx] = d_bias_val;
    }
}

__global__ void global_avg_pool_backward_kernel(
    const float *d_output,
    float *d_input,
    int batch_size,
    int sequence_length,
    int n_filters
) {
    int batch_idx = blockIdx.x;
    int filter_idx = threadIdx.x;
    
    if (filter_idx < n_filters) {
        float d_val = d_output[batch_idx * n_filters + filter_idx] / sequence_length;
        for (int s = 0; s < sequence_length; s++) {
            d_input[batch_idx * sequence_length * n_filters + 
                   s * n_filters + filter_idx] = d_val;
        }
    }
}

__global__ void relu_backward_kernel(
    const float *d_output,
    const float *input,
    float *d_input,
    int total_elements
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_elements) {
        d_input[idx] = input[idx] > 0 ? d_output[idx] : 0;
    }
}

__global__ void conv1d_backward_kernel(
    const float *d_output,
    const float *input,
    const float *weights,
    float *d_input,
    float *d_weights,
    float *d_bias,
    int batch_size,
    int sequence_length,
    int n_inputs,
    int n_filters,
    int kernel_size
) {
    int batch_idx = blockIdx.x;
    int filter_idx = blockIdx.y;
    int seq_idx = threadIdx.x;
    
    if (seq_idx < sequence_length) {
        if (batch_idx == 0 && seq_idx == 0) {
            float d_bias_val = 0.0f;
            for (int b = 0; b < batch_size; b++) {
                for (int s = 0; s < sequence_length; s++) {
                    d_bias_val += d_output[b * sequence_length * n_filters +
                                         s * n_filters + filter_idx];
                }
            }
            d_bias[filter_idx] = d_bias_val;
        }
        
        for (int k = 0; k < kernel_size; k++) {
            int seq_pos = seq_idx - kernel_size/2 + k;
            if (seq_pos >= 0 && seq_pos < sequence_length) {
                for (int c = 0; c < n_inputs; c++) {
                    float d_output_val = d_output[
                        batch_idx * sequence_length * n_filters +
                        seq_idx * n_filters + filter_idx
                    ];
                    
                    atomicAdd(&d_input[
                        batch_idx * sequence_length * n_inputs +
                        seq_pos * n_inputs + c
                    ], d_output_val * weights[
                        filter_idx * n_inputs * kernel_size +
                        c * kernel_size + k
                    ]);
                    
                    atomicAdd(&d_weights[
                        filter_idx * n_inputs * kernel_size +
                        c * kernel_size + k
                    ], d_output_val * input[
                        batch_idx * sequence_length * n_inputs +
                        seq_pos * n_inputs + c
                    ]);
                }
            }
        }
    }
}

__global__ void clip_gradients_kernel(float *grads, int size, float max_norm) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float grad = grads[idx];
        if (grad > max_norm) {
            grads[idx] = max_norm;
        } else if (grad < -max_norm) {
            grads[idx] = -max_norm;
        }
    }
}

__global__ void update_parameters_kernel(
    float *params,
    const float *grads,
    int size,
    float learning_rate
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        params[idx] -= learning_rate * grads[idx];
    }
}

static void forward_pass(
    Model *model,
    Activations *acts,
    const float *input,
    int batch_size,
    bool is_training
) {
    CHECK_CUDA(hipMemcpy(acts->conv_inputs, input,
        batch_size * model->sequence_length * model->n_inputs * sizeof(float),
        hipMemcpyHostToDevice));
    
    float *layer_input = acts->conv_inputs;
    
    for (int i = 0; i < model->n_conv_layers; i++) {
        ConvLayer *layer = &model->conv_layers[i];
        int in_channels = (i == 0) ? model->n_inputs : model->n_filters;
        
        if (i > 0) {
            CHECK_CUDA(hipMemcpy(acts->residual_outputs, layer_input,
                batch_size * model->sequence_length * model->n_filters * sizeof(float),
                hipMemcpyDeviceToDevice));
        }
        
        dim3 conv_blocks(batch_size, model->n_filters);
        dim3 conv_threads(model->sequence_length);
        conv1d_forward_kernel<<<conv_blocks, conv_threads>>>(
            layer_input,
            acts->conv_outputs,
            layer->weights,
            layer->bias,
            batch_size,
            model->sequence_length,
            in_channels,
            model->n_filters,
            model->kernel_size
        );
        
        // RMSNorm
        dim3 norm_blocks(batch_size);
        dim3 norm_threads(model->sequence_length);
        rms_norm_forward_kernel<<<norm_blocks, norm_threads>>>(
            acts->conv_outputs,
            acts->norm_outputs,
            batch_size,
            model->sequence_length,
            model->n_filters
        );
        
        int total_elements = batch_size * model->sequence_length * model->n_filters;
        int block_size = 256;
        int num_blocks = (total_elements + block_size - 1) / block_size;
        
        relu_forward_kernel<<<num_blocks, block_size>>>(
            acts->norm_outputs,
            acts->relu_outputs,
            batch_size,
            model->sequence_length,
            model->n_filters
        );
        
        if (i > 0) {
            residual_add_kernel<<<num_blocks, block_size>>>(
                acts->relu_outputs,
                acts->residual_outputs,
                batch_size,
                model->sequence_length,
                model->n_filters
            );
        }
        
        layer_input = acts->relu_outputs;
    }
    
    global_avg_pool_kernel<<<batch_size, model->n_filters>>>(
        layer_input,
        acts->pooled_output,
        batch_size,
        model->sequence_length,
        model->n_filters
    );
    
    dense_forward_kernel<<<batch_size, model->n_outputs>>>(
        acts->pooled_output,
        acts->final_output,
        model->dense_layer.weights,
        model->dense_layer.bias,
        batch_size,
        model->n_filters,
        model->n_outputs
    );
}

static void backward_pass(
    Model *model,
    Activations *acts,
    Gradients *grads,
    const float *targets,
    int batch_size
) {
    float *d_layer_output = grads->d_pooled_output;
    
    dense_backward_kernel<<<batch_size, model->n_filters>>>(
        grads->d_final_output,
        acts->pooled_output,
        model->dense_layer.weights,
        grads->d_pooled_output,
        model->dense_layer.d_weights,
        model->dense_layer.d_bias,
        batch_size,
        model->n_filters,
        model->n_outputs
    );
    
    for (int i = model->n_conv_layers - 1; i >= 0; i--) {
        ConvLayer *layer = &model->conv_layers[i];
        int in_channels = (i == 0) ? model->n_inputs : model->n_filters;
        
        if (i == model->n_conv_layers - 1) {
            global_avg_pool_backward_kernel<<<batch_size, model->n_filters>>>(
                d_layer_output,
                grads->d_relu_outputs,
                batch_size,
                model->sequence_length,
                model->n_filters
            );
            d_layer_output = grads->d_relu_outputs;
        }
        
        int total_elements = batch_size * model->sequence_length * model->n_filters;
        int block_size = 256;
        int num_blocks = (total_elements + block_size - 1) / block_size;
        
        relu_backward_kernel<<<num_blocks, block_size>>>(
            d_layer_output,
            acts->norm_outputs,
            grads->d_norm_outputs,
            total_elements
        );
        
        dim3 norm_blocks(batch_size);
        dim3 norm_threads(model->sequence_length);
        rms_norm_backward_kernel<<<norm_blocks, norm_threads>>>(
            grads->d_norm_outputs,
            acts->conv_outputs,
            grads->d_conv_outputs,
            batch_size,
            model->sequence_length,
            model->n_filters
        );
        
        dim3 conv_blocks(batch_size, model->n_filters);
        dim3 conv_threads(model->sequence_length);
        conv1d_backward_kernel<<<conv_blocks, conv_threads>>>(
            grads->d_conv_outputs,
            acts->conv_inputs,
            layer->weights,
            grads->d_conv_inputs,
            layer->d_weights,
            layer->d_bias,
            batch_size,
            model->sequence_length,
            in_channels,
            model->n_filters,
            model->kernel_size
        );
        
        d_layer_output = grads->d_conv_inputs;
    }
}

static void update_parameters(Model *model, float learning_rate) {
    int block_size = 256;
    
    for (int i = 0; i < model->n_conv_layers; i++) {
        ConvLayer *layer = &model->conv_layers[i];
        int in_channels = (i == 0) ? model->n_inputs : model->n_filters;
        
        int weights_size = model->n_filters * in_channels * model->kernel_size;
        int num_blocks = (weights_size + block_size - 1) / block_size;
        
        clip_gradients_kernel<<<num_blocks, block_size>>>(
            layer->d_weights, weights_size, MAX_GRAD_NORM);
        update_parameters_kernel<<<num_blocks, block_size>>>(
            layer->weights, layer->d_weights, weights_size, learning_rate);
        
        num_blocks = (model->n_filters + block_size - 1) / block_size;
        clip_gradients_kernel<<<num_blocks, block_size>>>(
            layer->d_bias, model->n_filters, MAX_GRAD_NORM);
        update_parameters_kernel<<<num_blocks, block_size>>>(
            layer->bias, layer->d_bias, model->n_filters, learning_rate);
    }
    
    int dense_weights_size = model->n_filters * model->n_outputs;
    int num_blocks = (dense_weights_size + block_size - 1) / block_size;
    
    clip_gradients_kernel<<<num_blocks, block_size>>>(
        model->dense_layer.d_weights, dense_weights_size, MAX_GRAD_NORM);
    update_parameters_kernel<<<num_blocks, block_size>>>(
        model->dense_layer.weights, model->dense_layer.d_weights,
        dense_weights_size, learning_rate);
    
    num_blocks = (model->n_outputs + block_size - 1) / block_size;
    clip_gradients_kernel<<<num_blocks, block_size>>>(
        model->dense_layer.d_bias, model->n_outputs, MAX_GRAD_NORM);
    update_parameters_kernel<<<num_blocks, block_size>>>(
        model->dense_layer.bias, model->dense_layer.d_bias,
        model->n_outputs, learning_rate);
}

__global__ void mse_loss_kernel(
    const float *predictions,
    const float *targets,
    float *loss,
    float *d_predictions,
    int batch_size,
    int n_outputs
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size * n_outputs) {
        float diff = predictions[idx] - targets[idx];
        d_predictions[idx] = 2.0f * diff / (batch_size * n_outputs);
        atomicAdd(loss, diff * diff / (batch_size * n_outputs));
    }
}

static void train(
    Model *model,
    Dataset *data,
    int n_epochs,
    int batch_size,
    float learning_rate
) {
    Activations *acts = create_activations(model, batch_size);
    Gradients *grads = create_gradients(model, batch_size);
    
    float *d_batch_inputs, *d_batch_targets, *d_loss;
    int batch_input_size = batch_size * model->sequence_length * model->n_inputs;
    int batch_target_size = batch_size * model->n_outputs;
    
    CHECK_CUDA(hipMalloc(&d_batch_inputs, batch_input_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_batch_targets, batch_target_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_loss, sizeof(float)));
    
    int n_batches = data->n_sequences / batch_size;
    for (int epoch = 0; epoch < n_epochs; epoch++) {
        float total_loss = 0.0f;
        
        for (int batch = 0; batch < n_batches; batch++) {
            float *batch_inputs = (float*)malloc(batch_input_size * sizeof(float));
            float *batch_targets = (float*)malloc(batch_target_size * sizeof(float));
            
            for (int i = 0; i < batch_size; i++) {
                int seq_idx = batch * batch_size + i;
                
                for (int t = 0; t < model->sequence_length; t++) {
                    for (int f = 0; f < model->n_inputs; f++) {
                        batch_inputs[i * model->sequence_length * model->n_inputs +
                                   t * model->n_inputs + f] = 
                            data->inputs[seq_idx][t][f];
                    }
                }
                
                for (int f = 0; f < model->n_outputs; f++) {
                    batch_targets[i * model->n_outputs + f] = 
                        data->targets[seq_idx][f];
                }
            }
            
            CHECK_CUDA(hipMemcpy(d_batch_inputs, batch_inputs,
                batch_input_size * sizeof(float), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_batch_targets, batch_targets,
                batch_target_size * sizeof(float), hipMemcpyHostToDevice));
            
            forward_pass(model, acts, d_batch_inputs, batch_size, true);
            
            CHECK_CUDA(hipMemset(d_loss, 0, sizeof(float)));
            int total_elements = batch_size * model->n_outputs;
            int block_size = 256;
            int num_blocks = (total_elements + block_size - 1) / block_size;
            
            mse_loss_kernel<<<num_blocks, block_size>>>(
                acts->final_output,
                d_batch_targets,
                d_loss,
                grads->d_final_output,
                batch_size,
                model->n_outputs
            );
            
            backward_pass(model, acts, grads, d_batch_targets, batch_size);
            update_parameters(model, learning_rate);
            
            float batch_loss;
            CHECK_CUDA(hipMemcpy(&batch_loss, d_loss, sizeof(float),
                hipMemcpyDeviceToHost));
            total_loss += batch_loss;
            
            free(batch_inputs);
            free(batch_targets);
        }
        
        printf("Epoch %d/%d - Loss: %f\n", epoch + 1, n_epochs,
               total_loss / n_batches);
    }
    
    CHECK_CUDA(hipFree(d_batch_inputs));
    CHECK_CUDA(hipFree(d_batch_targets));
    CHECK_CUDA(hipFree(d_loss));
    free_activations(acts);
    free(grads);
}

static void free_model(Model *model) {
    for (int i = 0; i < model->n_conv_layers; i++) {
        ConvLayer *layer = &model->conv_layers[i];
        CHECK_CUDA(hipFree(layer->weights));
        CHECK_CUDA(hipFree(layer->bias));
        CHECK_CUDA(hipFree(layer->d_weights));
        CHECK_CUDA(hipFree(layer->d_bias));
    }
    
    free(model->conv_layers);
    
    CHECK_CUDA(hipFree(model->dense_layer.weights));
    CHECK_CUDA(hipFree(model->dense_layer.bias));
    CHECK_CUDA(hipFree(model->dense_layer.d_weights));
    CHECK_CUDA(hipFree(model->dense_layer.d_bias));
    
    free(model);
}

int main() {
    srand(time(NULL));
    
    Dataset* data = generate_data(1000, 32, 6, 4, 0.1);
    Model* model = create_model(data->sequence_length, data->n_inputs,
                              data->n_outputs);
    
    int n_epochs = 50;
    int batch_size = 32;
    float learning_rate = LEARNING_RATE;
    
    train(model, data, n_epochs, batch_size, learning_rate);
    
    time_t now = time(NULL);
    char fname[64];
    strftime(fname, sizeof(fname), "%Y%m%d_%H%M%S_data.csv",
             localtime(&now));
    save_csv(fname, data);
    printf("Data saved to: %s\n", fname);
    
    free_dataset(data);
    free_model(model);
    
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipDeviceReset());
    
    return 0;
}